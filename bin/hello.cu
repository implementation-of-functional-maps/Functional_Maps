#include "hip/hip_runtime.h"
#include <stdio.h>
    2	
    3	/* GPU用strcpy */
    4	__device__ void
    5	dev_strcpy(char *dst, const char *src)
    6	{
    7	    while (*dst++ = *src++);
    8	}
    9	
   10	/* GPU側エントリ */
   11	__global__ void gen_hello(char *A)
   12	{
   13	    dev_strcpy(A, "Hello, World!");
   14	}
   15	
   16	int main()
   17	{
   18	    char *d_hello;
   19	
   20	    /* ホストのメモリを確保 */
   21	    char hello[128];
   22	
   23	    /* GPU側のメモリ(デバイスメモリ)確保 */
   24	    hipMalloc((void**)&d_hello, 128);
   25	
   26	    /* gen_hello 呼び出し */
   27	    gen_hello<<<1,1>>>(d_hello);
   28	
   29	    /* GPU側のデータを取得 */
   30	    hipMemcpy(hello, d_hello, 128, hipMemcpyDeviceToHost);
   31	
   32	    /* 確保したメモリを解放 */
   33	    hipFree(d_hello);
   34	
   35	    /* 出力 */
   36	    puts(hello);
   37	}